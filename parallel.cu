
#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCK_SIZE 256

__global__ void calculateNext(double* oldCylinder, double* newCylinder, const unsigned long long int numSlices, const unsigned long long int totalTime) {
    int i = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    if (i < numSlices) {
        if (i == 0) {
            newCylinder[i] = (oldCylinder[i] + oldCylinder[i + 1]) / 2.0;
        } else if (i == numSlices - 1) {
            newCylinder[i] = (oldCylinder[i - 1] + oldCylinder[i]) / 2.0;
        } else {
            newCylinder[i] = (oldCylinder[i - 1] + oldCylinder[i + 1]) / 2.0;
        }
    }
}

__global__ void initializeArray(double* cylinder, const unsigned long long int numSlices, const double concentration) {
    int i = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    if (i < numSlices) {
        if (i == 0) {
            cylinder[i] = concentration;
        } else {
            cylinder[i] = 0.0;
        }
    }
}

extern "C" double gpuCalculate(const unsigned long long int numSlices, const unsigned long long int totalTime, const double concentration, const unsigned long long int desiredPoint) {
    hipError_t mallocResult;
    double* oldCylinder;
    double* newCylinder;
    double* temp;

    mallocResult = hipMalloc((void**) &oldCylinder, numSlices * sizeof(double));
    if (mallocResult != hipSuccess) {
        fprintf(stderr, "CUDA Malloc failed, exiting...\n");
        exit(EXIT_FAILURE);
    }

    mallocResult = hipMalloc((void**) &newCylinder, numSlices * sizeof(double));
    if (mallocResult != hipSuccess) {
        fprintf(stderr, "CUDA Malloc failed, exiting...\n");
        exit(EXIT_FAILURE);
    }

    dim3 dimBlock(BLOCK_SIZE);
    unsigned long long int gridSize = ceil(numSlices / (double) BLOCK_SIZE);
    dim3 dimGrid(gridSize);

    initializeArray<<<dimGrid, dimBlock>>>(oldCylinder, numSlices, concentration);

    for (int i = 0; i < totalTime; i++) {
        calculateNext<<<dimGrid, dimBlock>>>(oldCylinder, newCylinder, numSlices, totalTime);
        temp = oldCylinder;
        oldCylinder = newCylinder;
        newCylinder = temp;
    }

    double answer;
    mallocResult = hipMemcpy(&answer, &oldCylinder[desiredPoint], sizeof(double), hipMemcpyDeviceToHost);
    if (mallocResult != hipSuccess) {
        fprintf(stderr, "CUDA Memcpy failed, exiting...\n");
        exit(EXIT_FAILURE);
    }

    mallocResult = hipFree(oldCylinder);
    if (mallocResult != hipSuccess) {
        fprintf(stderr, "CUDA free failed, exiting...\n");
        exit(EXIT_FAILURE);
    }

    mallocResult = hipFree(newCylinder);
    if (mallocResult != hipSuccess) {
        fprintf(stderr, "CUDA free failed, exiting...\n");
        exit(EXIT_FAILURE);
    }

    return answer;
}